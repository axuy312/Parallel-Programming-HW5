#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void mandelKernel(int* device_img, int width, float stepX, float stepY, float lowerX, float lowerY, int count) {
    // To avoid error caused by the floating number, use the following pseudo code
    //
	
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int thisX = idx % width;
    int thisY = idx / width;
		
    float c_re = lowerX + thisX * stepX;
    float c_im = lowerY + thisY * stepY;
	
	float z_re = c_re, z_im = c_im;
	int i;
	for (i = 0; i < count; ++i)
	{
		float z_re2 = z_re * z_re;
		float z_im2 = z_im * z_im;
		if (z_re2 + z_im2 > 4.f)
			break;

		float new_re = z_re2 - z_im2;
		float new_im = 2.f * z_re * z_im;
		z_re = c_re + new_re;
		z_im = c_im + new_im;
	}

    device_img[idx] = i;
	
	//printf("(%d,%d) => %d\n", thisX, thisY, i);
}

// Host front-end function that allocates the memory and launches the GPU kernel
void hostFE (float upperX, float upperY, float lowerX, float lowerY, int* img, int resX, int resY, int maxIterations)
{
    float stepX = (upperX - lowerX) / resX;
    float stepY = (upperY - lowerY) / resY;
    int size = resX * resY * sizeof(int);
	
    int *host_img;
    host_img = (int*) malloc(size);
	
	int *device_img;
    hipMalloc(&device_img, size);

    int numThreads = 800;
    int numBlocks = resX * resY / numThreads;
    mandelKernel<<<numBlocks, numThreads>>>(device_img, resX, stepX, stepY, lowerX, lowerY, maxIterations);
	
    hipMemcpy(host_img, device_img, size, hipMemcpyDeviceToHost);
	memcpy(img, host_img, size);
	
    hipFree(device_img);
    free(host_img);
}
